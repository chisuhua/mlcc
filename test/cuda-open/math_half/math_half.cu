#include "hip/hip_runtime.h"
// MIT License
//
// Copyright (c) 2018 Advanced Micro Devices, Inc. All Rights Reserved.
//
// Permission is hereby granted, free of charge, to any person
// obtaining a copy of this software and associated documentation
// files (the "Software"), to deal in the Software without
// restriction, including without limitation the rights to use, copy,
// modify, merge, publish, distribute, sublicense, and/or sell copies
// of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be
// included in all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
// EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
// MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
// NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS
// BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN
// ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
// CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

// These test only check if the code compiles, we don't test
// functionality yet.
// Reference: Cuda Toolkit v 9.2.88
//  1.1 Half Precision Intrinsics

#include <stdio.h>
#include <hip/hip_host_runtime_api.h>
#include <hip/hip_fp16.h>
#define N 10

__global__
void testHalfMath(__half *b)
{
  int i = blockIdx.x;
  unsigned u = (unsigned) i;
  __half h = (__half) i;
  if (i<N) {
    // 1.1 Half Precision Intrinsics

    // 1.1.1 Half Arithmetic Functions
    // b[i] = __hadd(h,h); // Fixme: missing functions __nv_hadd
    // b[i] = __hadd_sat(h,h); // Fixme: missing functions __nv_hadd_sat
    //  b[i] = __hdiv(h,h); // Fixme: Add __hdiv to cuda_open_headers
    b[i] = __hadd(h, h);
  }
}

void printArray(__half *array)
{
  printf("[");
  bool first = true;
  for (int i = 0; i<N; ++i)
  {
    if (first)
    {
      printf("%d", (int)array[i]);
      first = false;
    }
    else
    {
      printf(", %d", (int)array[i]);
    }
  }
  printf("]");
}

void printHipError(hipError_t error)
{
  printf("Hip Error: %s\n", hipGetErrorString(error));
}

bool hipCallSuccessful(hipError_t error)
{
  if (error != hipSuccess)
    printHipError(error);
  return error == hipSuccess;
}

bool deviceCanCompute(int deviceID)
{
  bool canCompute = false;
  hipDeviceProp_t deviceProp;
  bool devicePropIsAvailable =
    hipCallSuccessful(hipGetDeviceProperties(&deviceProp, deviceID));
  if (devicePropIsAvailable)
  {
    canCompute = deviceProp.computeMode != hipComputeModeProhibited;
    if (!canCompute)
      printf("Compute mode is prohibited\n");
  }
  return canCompute;
}

bool deviceIsAvailable(int *deviceID)
{
  return hipCallSuccessful(hipGetDevice(deviceID));
}

// We always use device 0
bool haveComputeDevice()
{
  int deviceID = 0;
  return deviceIsAvailable(&deviceID) && deviceCanCompute(deviceID);
}

int main()
{

  __half hostArray[N];

  if (!haveComputeDevice())
  {
    printf("No compute device available\n");
    return 0;
  }

  for (int i = 0; i<N; ++i)
    hostArray[i] = 0;

  printf("Array content before kernel:\n");
  printArray(hostArray);
  printf("\n");

  __half *deviceArray;
  if (!hipCallSuccessful(hipMalloc((void **)&deviceArray, N*sizeof(int))))
  {
    printf("Unable to allocate device memory\n");
    return 0;
  }

  hipLaunchKernelGGL((testHalfMath), dim3(N), dim3(1), 0, 0, deviceArray);

  if (hipCallSuccessful(hipMemcpy(hostArray,
                                     deviceArray,
                                     N * sizeof(__half),
                                     hipMemcpyDeviceToHost)))
  {
    printf("Array content after kernel:\n");
    printArray(hostArray);
    printf("\n");
  }
  else
  {
    printf("Unable to copy memory from device to host\n");
  }

  hipFree(deviceArray);
  return 0;
}
